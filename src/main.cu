#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define N 1000000
#define MAX_ERR 1e-6

class Test
{
private:
    int a;
public:
    Test()
    {
        a = 3;
    }
    void Print()
    {
        printf("class test %d \n", a);
    }

};

struct StructTest
{
    int a = 0;
    int b = 1;
};


__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

__global__ void vector_add_plain(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int last_index = n - 1;
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i > last_index)
        return;

    out[i] = a[i] + b[i];    
}

int main()
{
    //gridDim()    
    dim3 block_dim(512, 1, 1);
    int num_blocks = ceil(N / (float)block_dim.x);
    printf("a: %f \n", N / (float)block_dim.x);
    dim3 grid_dim(num_blocks, 1, 1);

    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    vector_add<<<grid_dim, block_dim>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    printf("Test \n");    
    //std::cout << "test" << "\n";
    std::cout <<out[0] << "\n";

    // Verification
    for(int i = 0; i < N; i++){
        bool is_ok = fabs(out[i] - a[i] - b[i]) < MAX_ERR;
        if (!is_ok)
        {
            printf("error!, i: %d \n", i);
            break;
        }
        //assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    
    printf("out[0] = %f\n", out[0]);
    std::cout << "passed" << "\n";    

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);

    return 0;
}