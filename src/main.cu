
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

class Test
{
private:
    int a;
public:
    Test()
    {
        a = 3;
    }
    void Print()
    {
        printf("class test %d \n", a);
    }

};

struct StructTest
{
    int a = 0;
    int b = 1;
};

int main()
{
    //gridDim()
    dim3 grid_dim(8, 1, 1);
    dim3 block_dim(512, 1, 1);
        
    //helloCUDA<<<grid_dim, block_dim>>>();
    helloCUDA<<<5, 1>>>();
    // Test test;
    // test.Print();

    // StructTest st;
    // printf("struct test: %d \n", st.b);

    hipDeviceSynchronize();        

    return 0;
}